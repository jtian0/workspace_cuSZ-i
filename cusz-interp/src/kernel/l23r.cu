/**
 * @file l23r.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.4
 * @date 2023-04-04
 *
 * (C) 2023 by Indiana University, Argonne National Laboratory
 *
 */

#include <hip/hip_runtime.h>

#include <type_traits>

// deps
#include "cusz/type.h"
#include "kernel/lrz.hh"
#include "mem/compact.hh"
#include "utils/err.hh"
#include "utils/timer.hh"
// definitions
#include "detail/l23r.cu_hip.inl"

template <typename T, typename Eq, bool ZigZag>
pszerror psz_comp_l23r(
    T* const data, dim3 const len3, f8 const eb, int const radius,
    Eq* const eq, void* _outlier, f4* time_elapsed, void* stream)
{
  static_assert(
      std::is_same<Eq, u4>::value or std::is_same<Eq, uint16_t>::value or
          std::is_same<Eq, uint8_t>::value,
      "Eq must be unsigned integer that is less than or equal to 4 bytes.");

  auto divide3 = [](dim3 len, dim3 tile) {
    return dim3(
        (len.x - 1) / tile.x + 1, (len.y - 1) / tile.y + 1,
        (len.z - 1) / tile.z + 1);
  };

  auto ndim = [&]() {
    if (len3.z == 1 and len3.y == 1)
      return 1;
    else if (len3.z == 1 and len3.y != 1)
      return 2;
    else
      return 3;
  };

  using Compact = typename CompactDram<PROPER_GPU_BACKEND, T>::Compact;

  auto ot = (Compact*)_outlier;

  constexpr auto Tile1D = 256;
  constexpr auto Seq1D = 4;
  constexpr auto Block1D = 64;
  auto Grid1D = divide3(len3, Tile1D);

  constexpr auto Tile2D = dim3(16, 16, 1);
  constexpr auto Block2D = dim3(16, 2, 1);
  auto Grid2D = divide3(len3, Tile2D);

  constexpr auto Tile3D = dim3(32, 8, 8);
  constexpr auto Block3D = dim3(32, 8, 1);
  auto Grid3D = divide3(len3, Tile3D);

  auto d = ndim();

  // error bound
  auto ebx2 = eb * 2;
  auto ebx2_r = 1 / ebx2;
  auto leap3 = dim3(1, len3.x, len3.x * len3.y);

  CREATE_GPUEVENT_PAIR;
  START_GPUEVENT_RECORDING((hipStream_t)stream);

  if (d == 1) {
    psz::rolling::c_lorenzo_1d1l<T, Eq, T, Tile1D, Seq1D>
        <<<Grid1D, Block1D, 0, (hipStream_t)stream>>>(
            data, len3, leap3, radius, ebx2_r, eq, ot->val(), ot->idx(),
            ot->num());
  }
  else if (d == 2) {
    psz::rolling::c_lorenzo_2d1l<T, Eq, T>
        <<<Grid2D, Block2D, 0, (hipStream_t)stream>>>(
            data, len3, leap3, radius, ebx2_r, eq, ot->val(), ot->idx(),
            ot->num());
  }
  else if (d == 3) {
    psz::rolling::c_lorenzo_3d1l<T, Eq, T>
        <<<Grid3D, Block3D, 0, (hipStream_t)stream>>>(
            data, len3, leap3, radius, ebx2_r, eq, ot->val(), ot->idx(),
            ot->num());
  }

  STOP_GPUEVENT_RECORDING((hipStream_t)stream);
  CHECK_GPU(hipStreamSynchronize((hipStream_t)stream));
  TIME_ELAPSED_GPUEVENT(time_elapsed);
  DESTROY_GPUEVENT_PAIR;

  return CUSZ_SUCCESS;
}

#define INIT(T, E, ZIGZAG)                                           \
  template pszerror psz_comp_l23r<T, E, ZIGZAG>(                     \
      T* const data, dim3 const len3, f8 const eb, int const radius, \
      E* const eq, void* _outlier, f4* time_elapsed, void* stream);

INIT(f4, u4, false)
INIT(f4, u4, true)
INIT(f8, u4, false)
INIT(f8, u4, true)

#undef INIT